#include "hip/hip_runtime.h"
#include "FluidTest2.cuh"

#include "systems/ParticleFluid2.cuh"

#include <imgui.h>
#include <thrust/extrema.h>

#include <iostream>

#define CUDA_CHECK(call)                                               \
  do                                                                   \
  {                                                                    \
    hipError_t error = call;                                          \
    if (error != hipSuccess)                                          \
    {                                                                  \
      fprintf(stderr, "CUDA error at %s:%d: %s\n", __FILE__, __LINE__, \
              hipGetErrorString(error));                              \
      exit(EXIT_FAILURE);                                              \
    }                                                                  \
  } while (0)

FluidTest2::FluidTest2(Game &game) : DefaultScreen(game),
                                    //  density_data(tex_size.x * tex_size.y),
                                     density_texture_data(tex_size.x * tex_size.y * 4)
{
}

// __global__ void density_to_texture(float *density_data, unsigned char *density_texture_data, int size, float max_density)

void check_cuda(const std::string &msg)
{
  hipError_t err = hipGetLastError();
  if (err != hipSuccess)
  {
    std::cerr << "FluidTest2: " << msg << ": " << hipGetErrorString(err) << std::endl;
  }
}

void FluidTest2::render(float _dt)
{
  render_start();
  fluid.update();
  // TODO: calculate expected max_density from particles per cell
  fluid.calculate_density_grid(density_texture_data, tex_size.x, tex_size.y, 300.0f);

  hipArray *cuda_array = density_renderer.cuda_map_texture();
  if (cuda_array == nullptr)
  {
    std::cerr << "Failed to map texture to CUDA" << std::endl;
    return;
  }

  hipDeviceSynchronize();
  density_renderer.update_texture_from_cuda(density_texture_data.data().get());
  check_cuda("update_texture_from_cuda");

  density_renderer.cuda_unmap_texture();

  density_renderer.set_transform(vp.get_transform());
  density_renderer.begin();
  for (int x_offset = -1; x_offset <= 1; ++x_offset)
  {
    for (int y_offset = -1; y_offset <= 1; ++y_offset)
    {
      float x_offset_f = x_offset * bounds.x;
      float y_offset_f = y_offset * bounds.y;
      density_renderer.add_rect(x_offset_f, y_offset_f, bounds.x, bounds.y, glm::vec3(1.0f));
    }
  }
  density_renderer.end();
  density_renderer.render();

  fluid.render(vp.get_transform());
  check_cuda("fluid.render");

  render_end();
  check_cuda("render_end");
}