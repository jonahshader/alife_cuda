#include "hip/hip_runtime.h"
#include "FluidSoil.cuh"

#include "systems/ParticleFluid2.cuh"
#include "systems/TimingProfiler.cuh"

#include <imgui.h>
#include <thrust/extrema.h>

#include <iostream>


#define CUDA_CHECK(call)                                                                           \
  do {                                                                                             \
    hipError_t error = call;                                                                      \
    if (error != hipSuccess) {                                                                    \
      fprintf(stderr, "CUDA error at %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(error)); \
      exit(EXIT_FAILURE);                                                                          \
    }                                                                                              \
  } while (0)

FluidSoil::FluidSoil(Game &game)
    : DefaultScreen(game),
      //  density_data(tex_size.x * tex_size.y),
      density_texture_data(tex_size.x * tex_size.y * 4) {}

bool FluidSoil::handleInput(SDL_Event event) {
  if (DefaultScreen::handleInput(event))
    return true;

  if (event.type == SDL_MOUSEBUTTONDOWN) {
    if (event.button.button == SDL_BUTTON_LEFT) {
      grabbing = true;
      return true;
    } else if (event.button.button == SDL_BUTTON_RIGHT) {
      repelling = true;
      return true;
    }
  } else if (event.type == SDL_MOUSEBUTTONUP) {
    if (event.button.button == SDL_BUTTON_LEFT) {
      grabbing = false;
      return true;
    } else if (event.button.button == SDL_BUTTON_RIGHT) {
      repelling = false;
      return true;
    }
  } else if (event.type == SDL_MOUSEMOTION) {
    mouse_pos = {event.motion.x, event.motion.y};
    return true;
  } else if (event.type == SDL_KEYDOWN) {
    switch (event.key.keysym.sym) {
      case SDLK_PLUS:
      case SDLK_EQUALS:
      case SDLK_KP_PLUS:
      case SDLK_KP_EQUALS:
        grab_radius *= 1.25f;
        return true;
      case SDLK_MINUS:
      case SDLK_KP_MINUS:
        grab_radius *= 0.8f;
        return true;
      case SDLK_LEFTBRACKET:
        grab_strength *= 0.8f;
        return true;
      case SDLK_RIGHTBRACKET:
        grab_strength *= 1.25f;
        return true;
      default:
        break;
    }
  }

  return false;
}

// __global__ void density_to_texture(float *density_data, unsigned char *density_texture_data, int
// size, float max_density)

void FluidSoil::check_cuda(const std::string &msg) {
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    std::cerr << "FluidSoil: " << msg << ": " << hipGetErrorString(err) << std::endl;
  }
}

void FluidSoil::render(float _dt) {
  auto &profiler = TimingProfiler::getInstance();
  render_start();
  auto &circle_renderer = game.getResources().circle_renderer;
  auto &main_font_world = game.getResources().main_font_world;
  circle_renderer.set_transform(vp.get_transform());
  main_font_world.set_transform(vp.get_transform());
  circle_renderer.begin();
  main_font_world.begin();
  soil.update_cuda(_dt); // TODO: need proper dt parameter
  fluid.update(soil);

  ImGui::Begin("Particle Fluid");
  ImGui::Checkbox("Show Density Grid", &show_density_grid);
  ImGui::End();
  if (grabbing || repelling) {
    const auto world_coords = vp.unproject({mouse_pos.x, mouse_pos.y});

    fluid.attract({world_coords.x, world_coords.y}, repelling ? -grab_strength : grab_strength,
                  grab_radius);
  }
  // TODO: calculate expected max_density from particles per cell
  {
    auto scope = profiler.scopedMeasure("calculate_density_grid");
    if (show_density_grid)
      fluid.calculate_density_grid(density_texture_data, tex_size.x, tex_size.y, 300.0f);
  }

  {
    auto scope = profiler.scopedMeasure("density_renderer.cuda_map_texture()");
    if (show_density_grid) {
      hipArray *cuda_array = density_renderer.cuda_map_texture();
      if (cuda_array == nullptr) {
        std::cerr << "Failed to map texture to CUDA" << std::endl;
        return;
      }
    }
  }


  {
    auto scope = profiler.scopedMeasure("update_texture_from_cuda");
    if (show_density_grid) {
      density_renderer.update_texture_from_cuda(density_texture_data.data().get());
      check_cuda("update_texture_from_cuda");
    }
  }

  {
    auto scope = profiler.scopedMeasure("density_renderer.render...");
    if (show_density_grid) {
      density_renderer.cuda_unmap_texture();

      density_renderer.set_transform(vp.get_transform());
      density_renderer.begin();
      for (int x_offset = -1; x_offset <= 1; ++x_offset) {
        for (int y_offset = -1; y_offset <= 1; ++y_offset) {
          float x_offset_f = x_offset * bounds.x;
          float y_offset_f = y_offset * bounds.y;
          density_renderer.add_rect(x_offset_f, y_offset_f, bounds.x, bounds.y, glm::vec3(1.0f));
        }
      }
      density_renderer.end();
      density_renderer.render();
    }

  }

  {
    auto scope = profiler.scopedMeasure("soil.render");
    soil.render(vp.get_transform());
    check_cuda("soil.render");
  }

  {
    auto scope = profiler.scopedMeasure("fluid.render");
    fluid.render(vp.get_transform());
    check_cuda("fluid.render");
  }

  // add a circle for mouse grab tool
  float grab_color_opacity = 0.1f;
  if (grabbing || repelling) {
    grab_color_opacity = 0.25f;
  }
  const auto world_coords = vp.unproject({mouse_pos.x, mouse_pos.y});
  circle_renderer.add_circle(world_coords.x, world_coords.y, grab_radius,
                             glm::vec4(0.5f, 0.5f, 0.8f, grab_color_opacity));
  // display strength
  main_font_world.add_text(world_coords.x, world_coords.y, grab_radius,
                           "Strength: " + std::to_string(grab_strength),
                           glm::vec4(0.0f, 0.0f, 0.0f, 0.5f), FontRenderer::HAlign::CENTER);

  {
    auto scope = profiler.scopedMeasure("render_end...");

    circle_renderer.end();
    main_font_world.end();
    circle_renderer.render();
    main_font_world.render();

    render_end();
    check_cuda("render_end");
  }
  profiler_gui.render();
}
