#include "hip/hip_runtime.h"
//
// Created by jonah on 3/11/2024.
//

#include "TreeTest.cuh"
#include "glad/glad.h"

#include <iostream>
#include <chrono>

constexpr uint32_t NUM_NODES = 2<<7;
constexpr uint32_t NUM_TREES = 2<<8;

TreeBatch make_batch(uint32_t node_count, uint32_t tree_count, std::default_random_engine& rand) {
    std::vector<Tree> trees;
    for (int i = 0; i < NUM_TREES; ++i) {
        trees.push_back(build_tree_optimized(NUM_NODES, rand, glm::vec2(i * 64, 0)));
    }

    return concatenate_trees(trees);
}

TreeTest::TreeTest(Game &game) : game(game) {
    read_tree = make_batch(NUM_NODES, NUM_TREES, game.getResources().generator);
    write_tree = read_tree;
}

void TreeTest::show() {

}

void TreeTest::render(float dt) {
    glClearColor(0.0f, 0.0f, 0.0f, 1.0f);
    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
    auto& bold = game.getResources().extra_bold_font;
    auto& rect = game.getResources().rect_renderer;
    auto& line = game.getResources().line_renderer;
    bold.set_transform(vp.get_transform());
    rect.set_transform(vp.get_transform());
    line.set_transform(vp.get_transform());
    bold.begin();
    rect.begin();
    line.begin();

//    bold.add_text(0, (vp.get_height() / 2.0f) - 120, 600, "Tree Test", glm::vec4(0.5), FontRenderer::HAlign::CENTER);
//    rect.add_rect(0, 0, 32, 24, 8, glm::vec4(1));
//    rect.add_rect(0, -100, 128, 32, 16, glm::vec4(1, 0, 1, 1));
    auto start = std::chrono::steady_clock::now();
//    mutate(tree, game.getResources().generator, 0.002f);

// TODO: get mutation working again
//    if (!stripped_tree.empty()) {
//        mutate(stripped_tree, game.getResources().generator, 0.002f);
//    } else {
//        mutate(tree, game.getResources().generator, 0.002f);
//    }

//    if (mixing) {
//        mix_node_contents(read_tree, write_tree, 1.0f, total_energy);
//        read_tree.swap(write_tree);
//    }

    auto end = std::chrono::steady_clock::now();
    auto elapsed = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
    bold.add_text(0, (vp.get_height() / 2.0f), 500, "Mutate Time: " + std::to_string(elapsed.count()) + "us", glm::vec4(0.75), FontRenderer::HAlign::LEFT);

    start = std::chrono::steady_clock::now();
    update_tree(read_tree);
    end = std::chrono::steady_clock::now();
    elapsed = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
//    bold.add_text(0, (vp.get_height() / 2.0f) + 120, 500, "Update Time: " + std::to_string(elapsed.count()) + "us", glm::vec4(0.75), FontRenderer::HAlign::CENTER);

    render_tree(line, read_tree, game.getResources().generator);
//    std::cout << "Total energy: " << compute_total_energy(read_tree) << std::endl;
//    std::cout << "Min/max energy: " << get_min_energy(read_tree) << ',' << get_max_energy(read_tree) << std::endl;

//    bold.add_text(0, (vp.get_height() / 2.0f) + 240, 500, "Min/Max Energy: " + std::to_string(get_min_energy(read_tree)) + ',' + std::to_string(get_max_energy(read_tree)), glm::vec4(0.75), FontRenderer::HAlign::LEFT);


    bold.end();
    rect.end();
    line.end();

    bold.render();
    rect.render();
    line.render();

    SDL_GL_SwapWindow(game.getResources().window);
}

void TreeTest::resize(int width, int height) {
    vp.update(width, height);
}

void TreeTest::hide() {

}

void TreeTest::handleInput(SDL_Event event) {
    if (event.type == SDL_KEYDOWN) {
        if (event.key.keysym.sym == SDLK_ESCAPE) {
            game.stopGame();
        } else if (event.key.keysym.sym == SDLK_r) {
            read_tree = make_batch(NUM_NODES, NUM_TREES, game.getResources().generator);
            write_tree = read_tree;
        } else if (event.key.keysym.sym == SDLK_SPACE) {
            mixing = !mixing;
        }
    } else if (event.type == SDL_MOUSEWHEEL) {
        vp.handle_scroll(event.wheel.y);
    } else if (event.type == SDL_MOUSEMOTION) {
        if (event.motion.state & SDL_BUTTON_LMASK) {
            vp.handle_pan(event.motion.xrel, event.motion.yrel);
        }
    }
}