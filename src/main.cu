#include "hip/hip_runtime.h"
// https://bcmpinc.wordpress.com/2015/08/18/creating-an-opengl-4-5-context-using-sdl2-and-glad/

#define SDL_MAIN_HANDLED
//#define GLM_FORCE_CUDA
//#define GLM_COMPILER_CUDA
#include <glm/glm.hpp>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <thrust/random.h>
#include <iostream>
#include "systems/Game.cuh"
#include "screens/TreeTest.cuh"
#include "screens/FluidTest.cuh"
#include "screens/SoilTest.cuh"
#include "spatial_sort.cuh"

#include <SDL.h>
#include "glad/glad.h"
#include "imgui.h"
#include "imgui_impl_sdl2.h"
#include "imgui_impl_opengl3.h"

static int viewport_width = 1920;
static int viewport_height = 1080;
static SDL_Window* window = nullptr;
static SDL_GLContext main_context;

static void sdl_die(const char * message) {
    fprintf(stderr, "%s: %s\n", message, SDL_GetError());
    exit(2);
}

void init_screen(const char * caption) {
    // Initialize SDL
    if (SDL_Init(SDL_INIT_VIDEO) < 0)
        sdl_die("Couldn't initialize SDL");
    atexit (SDL_Quit);
    SDL_GL_LoadLibrary(nullptr); // Default OpenGL is fine.

    // Request an OpenGL 4.3 context (should be core)
    SDL_GL_SetAttribute(SDL_GL_ACCELERATED_VISUAL, 1);
    SDL_GL_SetAttribute(SDL_GL_CONTEXT_MAJOR_VERSION, 4);
    SDL_GL_SetAttribute(SDL_GL_CONTEXT_MINOR_VERSION, 3);

    // Also request a depth buffer
    SDL_GL_SetAttribute(SDL_GL_DOUBLEBUFFER, 1);
    SDL_GL_SetAttribute(SDL_GL_DEPTH_SIZE, 24);

#ifdef FULLSCREEN
    window = SDL_CreateWindow(
            caption,
            SDL_WINDOWPOS_UNDEFINED, SDL_WINDOWPOS_UNDEFINED,
            0, 0, SDL_WINDOW_FULLSCREEN_DESKTOP | SDL_WINDOW_OPENGL
    );
#else
    window = SDL_CreateWindow(
                caption,
                SDL_WINDOWPOS_UNDEFINED, SDL_WINDOWPOS_UNDEFINED,
                viewport_width, viewport_height, SDL_WINDOW_OPENGL | SDL_WINDOW_RESIZABLE
        );

#endif

    if (window == nullptr) sdl_die("Couldn't set video mode");

    main_context = SDL_GL_CreateContext(window);
    if (main_context == nullptr)
        sdl_die("Failed to create OpenGL context");

    // Check OpenGL properties
    printf("OpenGL loaded\n");
    gladLoadGLLoader(SDL_GL_GetProcAddress);
    printf("Vendor:   %s\n", glGetString(GL_VENDOR));
    printf("Renderer: %s\n", glGetString(GL_RENDERER));
    printf("Version:  %s\n", glGetString(GL_VERSION));

    // Use v-sync
    SDL_GL_SetSwapInterval(0);

    // Disable depth test and face culling.
    glDisable(GL_DEPTH_TEST);
    glDisable(GL_CULL_FACE);
    glEnable(GL_BLEND);
    glBlendFunc(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);

    int w,h;
    SDL_GetWindowSize(window, &w, &h);
    glViewport(0, 0, w, h);
    glClearColor(0.0f, 0.0f, 0.0f, 1.0f);
}

void init_imgui() {
    IMGUI_CHECKVERSION();
    ImGui::CreateContext();
    ImGuiIO& io = ImGui::GetIO();
    io.ConfigFlags |= ImGuiConfigFlags_NavEnableKeyboard;

    // setup platform/renderer backends
    ImGui_ImplSDL2_InitForOpenGL(window, main_context);
    ImGui_ImplOpenGL3_Init("#version 430"); // TODO: try without version
}

int main(int argc, char* argv[]) {
    hipDeviceProp_t cuda_prop;
    hipGetDeviceProperties(&cuda_prop, 0);
    // print the compute capability, max number of threads per block, max number of blocks, number of SMs, max number of threads per SM,
    // number of registers per block, number of registers per SM, shared memory per block, shared memory per SM, warp size, number of floating point units
    std::cout << "Compute capability: " << cuda_prop.major << "." << cuda_prop.minor << std::endl;
    std::cout << "Max threads per block: " << cuda_prop.maxThreadsPerBlock << std::endl;
    std::cout << "Max blocks: " << cuda_prop.maxGridSize[0] << std::endl;
    std::cout << "Number of SMs: " << cuda_prop.multiProcessorCount << std::endl;
    std::cout << "Max threads per SM: " << cuda_prop.maxThreadsPerMultiProcessor << std::endl;
    std::cout << "Number of registers per block: " << cuda_prop.regsPerBlock << std::endl;
    std::cout << "Number of registers per SM: " << cuda_prop.regsPerMultiprocessor << std::endl;
    std::cout << "Shared memory per block: " << cuda_prop.sharedMemPerBlock << " bytes" << std::endl;
    std::cout << "Shared memory per SM: " << cuda_prop.sharedMemPerMultiprocessor << " bytes" << std::endl;
    std::cout << "Warp size: " << cuda_prop.warpSize << std::endl;
    std::cout << "Number of floating point units: " << cuda_prop.multiProcessorCount * cuda_prop.maxThreadsPerMultiProcessor << std::endl;

    init_screen("OpenGL 4.3");
    init_imgui();

    //    jl_init();
    //
    //    jl_eval_string("print(sqrt(2.0))");

    Game game;
    game.getResources().window = window;
    game.resize(viewport_width, viewport_height);
    //    game.pushScreen(std::make_shared<MainMenu>(game));
    // game.pushScreen(std::make_shared<FluidTest>(game));
    // game.pushScreen(std::make_shared<TreeTest>(game));
    game.pushScreen(std::make_shared<SoilTest>(game));

    float time = 0;
    SDL_Event event;
    while (game.isRunning()) {
        while (SDL_PollEvent(&event)) {
            ImGui_ImplSDL2_ProcessEvent(&event);
            // skip game input handling if ImGui wants to capture the event
            if (!ImGui::GetIO().WantCaptureKeyboard && !ImGui::GetIO().WantCaptureMouse) {
                game.handleInput(event);
            }
            if (event.type == SDL_QUIT) {
                game.stopGame();
            } else if (event.type == SDL_WINDOWEVENT) {
                if (event.window.event == SDL_WINDOWEVENT_RESIZED) {
                    viewport_width = event.window.data1;
                    viewport_height = event.window.data2;
                    game.resize(viewport_width, viewport_height);
                    glViewport(0, 0, viewport_width, viewport_height);
                }
            }
        }

        // Start the Dear ImGui frame
        ImGui_ImplOpenGL3_NewFrame();
        ImGui_ImplSDL2_NewFrame();
        ImGui::NewFrame();

        game.render(1/165.0f);

        ImGui::Render();
        ImGui_ImplOpenGL3_RenderDrawData(ImGui::GetDrawData());
        SDL_GL_SwapWindow(game.getResources().window);
        time += 1/165.0f;
    }

    ImGui_ImplOpenGL3_Shutdown();
    ImGui_ImplSDL2_Shutdown();
    ImGui::DestroyContext();

    SDL_GL_DeleteContext(main_context);
    SDL_DestroyWindow(window);
    //    jl_atexit_hook(0);
    return 0;
}
