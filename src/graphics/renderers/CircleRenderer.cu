//
// Created by jonah on 4/18/2023.
//

#include "CircleRenderer.cuh"
#include <glad/glad.h>
#include <cuda_gl_interop.h>
#include <iostream>

CircleRenderer::CircleRenderer() : shader("shaders/circle.vert", "shaders/circle.frag")
{
    float baseMesh[] = {
        // t1
        -0.5f,
        -0.5f, // bottom left
        0.5f,
        -0.5f, // bottom right
        0.5f,
        0.5f, // top right
        // t2
        0.5f,
        0.5f,
        -0.5f,
        0.5f,
        -0.5f,
        -0.5f,
    };

    // create vao, buffers
    glGenVertexArrays(1, &vao);
    glGenBuffers(1, &vbo_data);
    glGenBuffers(1, &vbo_base_mesh);

    // buffer baseMesh
    glBindBuffer(GL_ARRAY_BUFFER, vbo_base_mesh);
    glBufferData(GL_ARRAY_BUFFER, sizeof(baseMesh), baseMesh, GL_STATIC_DRAW);
    glBindBuffer(GL_ARRAY_BUFFER, vbo_data);
    glBufferData(GL_ARRAY_BUFFER, 0, nullptr, GL_DYNAMIC_DRAW); // TODO: eval GL_DYNAMIC_DRAW

    glBindVertexArray(vao);
    glBindBuffer(GL_ARRAY_BUFFER, vbo_base_mesh);

    // x y
    glVertexAttribPointer(0, 2, GL_FLOAT, false, 2 * sizeof(float), (void *)0);

    // offset
    GLsizei s = 4 * sizeof(float);
    glBindBuffer(GL_ARRAY_BUFFER, vbo_data);
    glVertexAttribPointer(1, 2, GL_FLOAT, false, s, (void *)0); // x y sizes r g b
    glVertexAttribDivisor(1, 1);
    // size
    glVertexAttribPointer(2, 1, GL_FLOAT, false, s, (void *)(2 * sizeof(float)));
    glVertexAttribDivisor(2, 1);
    // color (rgba)
    glVertexAttribPointer(3, 4, GL_UNSIGNED_BYTE, true, s, (void *)(3 * sizeof(float)));
    glVertexAttribDivisor(3, 1);

    glEnableVertexAttribArray(0);
    glEnableVertexAttribArray(1);
    glEnableVertexAttribArray(2);
    glEnableVertexAttribArray(3);

    glBindBuffer(GL_ARRAY_BUFFER, 0);
    glBindVertexArray(0);
}

void CircleRenderer::begin()
{
    data.clear();
}

void CircleRenderer::end()
{
    glBindBuffer(GL_ARRAY_BUFFER, vbo_data);
    unsigned int data_bytes = data.size() * sizeof(data[0]);
    if (data_bytes > buffer_size)
    {
        // resize buffer
        buffer_size = data_bytes * 2;
        glBufferData(GL_ARRAY_BUFFER, buffer_size, nullptr, GL_DYNAMIC_DRAW);
        std::cout << "Doubled CircleRenderer buffer size from " << buffer_size / 2 << " to " << buffer_size << std::endl;
    }
    glBufferSubData(GL_ARRAY_BUFFER, 0, data_bytes, data.data());

    glBindBuffer(GL_ARRAY_BUFFER, 0);
}

void CircleRenderer::add_circle(float x, float y, float radius, glm::vec4 color)
{
    add_circle(x, y, radius, color.r * 255, color.g * 255, color.b * 255, 255);
}

void CircleRenderer::render()
{
    shader.use();
    glBindVertexArray(vao);
    glDrawArraysInstanced(GL_TRIANGLES, 0, 6, data.size() / ELEMS_PER_CIRCLE);
    glBindVertexArray(0);
}

CircleRenderer::~CircleRenderer()
{
    glDeleteVertexArrays(1, &vao);
    glDeleteBuffers(1, &vbo_base_mesh);
    glDeleteBuffers(1, &vbo_data);
}

void CircleRenderer::set_transform(glm::mat4 transform)
{
    shader.use();
    shader.setMatrix4("transform", transform);
}

void CircleRenderer::add_circle(float x, float y, float radius, unsigned char r, unsigned char g, unsigned char b,
                                unsigned char a)
{
    radius *= 2;
    data.emplace_back(reinterpret_cast<unsigned int &>(x));
    data.emplace_back(reinterpret_cast<unsigned int &>(y));
    data.emplace_back(reinterpret_cast<unsigned int &>(radius));
    // pack color into a single unsigned int
    unsigned int color = 0;
    color |= r;
    color |= g << 8;
    color |= b << 16;
    color |= a << 24;

    data.emplace_back(color);
}

void CircleRenderer::cuda_register_buffer()
{
    hipGraphicsGLRegisterBuffer(&cuda_resource, vbo_data, cudaGraphicsMapFlagsWriteDiscard);
}

void CircleRenderer::cuda_unregister_buffer()
{
    hipGraphicsUnregisterResource(cuda_resource);
}

void *CircleRenderer::cuda_map_buffer()
{
    void *device_ptr;
    size_t size;
    hipGraphicsMapResources(1, &cuda_resource, 0);
    hipGraphicsResourceGetMappedPointer(&device_ptr, &size, cuda_resource);
    return device_ptr;
}

void CircleRenderer::cuda_unmap_buffer()
{
    hipGraphicsUnmapResources(1, &cuda_resource, 0);
}

void CircleRenderer::ensure_vbo_capacity(size_t circles)
{
    const auto size_bytes = circles * CIRCLE_SIZE;
    if (buffer_size < size_bytes) {
        if (buffer_size == 0) {
            buffer_size = size_bytes;
        } else {
            while (buffer_size < size_bytes) {
                buffer_size *= 2;
            }
        }

        std::cout << "LineRenderer buffer size changed to " << buffer_size << std::endl;

        glBindBuffer(GL_ARRAY_BUFFER, vbo_data);
        glBufferData(GL_ARRAY_BUFFER, buffer_size, nullptr, GL_DYNAMIC_DRAW);
        glBindBuffer(GL_ARRAY_BUFFER, 0);
    } else if (buffer_size > size_bytes * 4) {
        buffer_size = size_bytes;
        glBindBuffer(GL_ARRAY_BUFFER, vbo_data);
        glBufferData(GL_ARRAY_BUFFER, buffer_size, nullptr, GL_DYNAMIC_DRAW);
        glBindBuffer(GL_ARRAY_BUFFER, 0);
    }
}