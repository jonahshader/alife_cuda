#include "SimpleRectRenderer.cuh"
#include <glad/glad.h>
#include <iostream>

#include <cuda_gl_interop.h>

SimpleRectRenderer::SimpleRectRenderer()
    : shader("shaders/rect_simple.vert", "shaders/rect_simple.frag") {
  // float base_mesh[] = {
  //     // t1
  //     -0.5f,
  //     -0.5f, // bottom left
  //     0.5f,
  //     -0.5f, // bottom right
  //     0.5f,
  //     0.5f, // top right
  //     // t2
  //     0.5f,
  //     0.5f,
  //     -0.5f,
  //     0.5f,
  //     -0.5f,
  //     -0.5f,
  // };

  float base_mesh[] = {
      // t1
      0.0f,
      0.0f, // bottom left
      1.0f,
      0.0f, // bottom right
      1.0f,
      1.0f, // top right
      // t2
      1.0f,
      1.0f,
      0.0f,
      1.0f,
      0.0f,
      0.0f,
  };

  // create vao, buffers
  glGenVertexArrays(1, &vao);
  glGenBuffers(1, &vbo_data);
  glGenBuffers(1, &vbo_base_mesh);

  // buffer baseMesh
  glBindBuffer(GL_ARRAY_BUFFER, vbo_base_mesh);
  glBufferData(GL_ARRAY_BUFFER, sizeof(base_mesh), base_mesh, GL_STATIC_DRAW);
  glBindBuffer(GL_ARRAY_BUFFER, vbo_data);
  glBufferData(GL_ARRAY_BUFFER, 0, nullptr, GL_STATIC_DRAW); // TODO: eval GL_DYNAMIC_DRAW

  glBindVertexArray(vao);
  glBindBuffer(GL_ARRAY_BUFFER, vbo_base_mesh);

  // x y
  glVertexAttribPointer(0, 2, GL_FLOAT, false, 2 * sizeof(float), (void *)0);

  // offset
  glBindBuffer(GL_ARRAY_BUFFER, vbo_data);
  glVertexAttribPointer(1, 2, GL_FLOAT, false, BYTES_PER_RECT, (void *)0); // x y sizes r g b
  glVertexAttribDivisor(1, 1);
  // size
  glVertexAttribPointer(2, 2, GL_FLOAT, false, BYTES_PER_RECT, (void *)(2 * sizeof(float)));
  glVertexAttribDivisor(2, 1);
  // color
  glVertexAttribPointer(3, 4, GL_FLOAT, false, BYTES_PER_RECT, (void *)(4 * sizeof(float)));
  glVertexAttribDivisor(3, 1);

  glEnableVertexAttribArray(0);
  glEnableVertexAttribArray(1);
  glEnableVertexAttribArray(2);
  glEnableVertexAttribArray(3);

  glBindBuffer(GL_ARRAY_BUFFER, 0);
  glBindVertexArray(0);
}

void SimpleRectRenderer::begin() {
  data.clear();
}

void SimpleRectRenderer::end() {
  glBindBuffer(GL_ARRAY_BUFFER, vbo_data);
  unsigned int data_bytes = data.size() * sizeof(data[0]);
  if (data_bytes > buffer_size) {
    // full update
    // current scheme: double buffer size
    // TODO: shrink buffer when data is less than half
    buffer_size = data_bytes * 2;
    glBufferData(GL_ARRAY_BUFFER, buffer_size, nullptr, GL_DYNAMIC_DRAW);
    std::cout << "Doubled SimpleRectRenderer buffer size from " << buffer_size / 2 << " to "
              << buffer_size << std::endl;
  }
  glBufferSubData(GL_ARRAY_BUFFER, 0, data_bytes, data.data());
  glBindBuffer(GL_ARRAY_BUFFER, 0);
}

void SimpleRectRenderer::add_rect(float x, float y, float width, float height, glm::vec4 color) {
  data.emplace_back(x);
  data.emplace_back(y);
  data.emplace_back(width);
  data.emplace_back(height);
  data.emplace_back(color.r);
  data.emplace_back(color.g);
  data.emplace_back(color.b);
  data.emplace_back(color.a);
}

void SimpleRectRenderer::render() {
  shader.use();
  glBindVertexArray(vao);
  glDrawArraysInstanced(GL_TRIANGLES, 0, 6, data.size() / FLOATS_PER_RECT);
  glBindVertexArray(0);
}

void SimpleRectRenderer::render(size_t rect_count) {
  shader.use();
  glBindVertexArray(vao);
  glDrawArraysInstanced(GL_TRIANGLES, 0, 6, rect_count);
  glBindVertexArray(0);
}

SimpleRectRenderer::~SimpleRectRenderer() {
  glDeleteVertexArrays(1, &vao);
  glDeleteBuffers(1, &vbo_base_mesh);
  glDeleteBuffers(1, &vbo_data);
}

void SimpleRectRenderer::set_transform(glm::mat4 transform) {
  shader.use();
  shader.setMatrix4("transform", transform);
}

void SimpleRectRenderer::cuda_register_buffer() {
  // cudaGraphicsMapFlagsWriteDiscard: cuda will only write, not read, to this buffer
  hipGraphicsGLRegisterBuffer(&cuda_resource, vbo_data, cudaGraphicsMapFlagsWriteDiscard);
}

void SimpleRectRenderer::cuda_unregister_buffer() {
  hipGraphicsUnregisterResource(cuda_resource);
}

void *SimpleRectRenderer::cuda_map_buffer() {
  void *device_ptr;
  size_t size;
  hipGraphicsMapResources(1, &cuda_resource, 0);
  hipGraphicsResourceGetMappedPointer(&device_ptr, &size, cuda_resource);
  return device_ptr;
}

void SimpleRectRenderer::cuda_unmap_buffer() {
  hipGraphicsUnmapResources(1, &cuda_resource, 0);
}

void SimpleRectRenderer::ensure_vbo_capacity(size_t num_rects) {
  const auto size_bytes = num_rects * BYTES_PER_RECT;
  if (buffer_size < size_bytes) {
    if (buffer_size == 0) {
      buffer_size = size_bytes;
    } else {
      while (buffer_size < size_bytes) {
        buffer_size *= 2;
      }
    }

    std::cout << "SimpleRectRenderer buffer size changed to " << buffer_size << std::endl;

    glBindBuffer(GL_ARRAY_BUFFER, vbo_data);
    glBufferData(GL_ARRAY_BUFFER, buffer_size, nullptr, GL_DYNAMIC_DRAW);
    glBindBuffer(GL_ARRAY_BUFFER, 0);
  } else if (buffer_size > size_bytes * 4) {
    buffer_size = size_bytes;
    glBindBuffer(GL_ARRAY_BUFFER, vbo_data);
    glBufferData(GL_ARRAY_BUFFER, buffer_size, nullptr, GL_DYNAMIC_DRAW);
    glBindBuffer(GL_ARRAY_BUFFER, 0);
  }
}
