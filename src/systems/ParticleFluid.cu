#include "hip/hip_runtime.h"
#include "ParticleFluid.cuh"

#include "Kernels.cuh"

constexpr float PRESSURE_MULTIPLIER = 1200000.0f;
constexpr float VISCOSITY_MULTIPLIER = 8.0f;
constexpr float TARGET_PRESSURE = 2.0f;
constexpr float PARTICLE_MASS = 1.0f;
constexpr float GRAVITY_ACCELERATION = 108.0;
constexpr float WALL_ACCEL_PER_DIST = 6600.0f;

__host__ __device__ int particle_to_gid(float x, float y, int grid_width, float cell_size)
{
  int grid_x = x / cell_size;
  int grid_y = y / cell_size;
  return grid_y * grid_width + grid_x;
}

__global__ void reset_particles_per_cell(int *particles_per_cell, int particles_per_cell_length)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x; // grid index
  if (i < particles_per_cell_length)
  {
    particles_per_cell[i] = 0;
  }
}

__global__ void populate_grid_indices(float *x_particle, float *y_particle, int *grid_indices, int max_particles,
                                      int *particles_per_cell, int max_particles_per_cell, int grid_width, int grid_height, float cell_size)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x; // particle id
  if (i >= max_particles)
  {
    return;
  }

  float x = x_particle[i];
  float y = y_particle[i];

  int grid_index = particle_to_gid(x, y, grid_width, cell_size);

  int slot_index = atomicAdd(&particles_per_cell[grid_index], 1);
  if (slot_index < max_particles_per_cell)
  {
    grid_indices[grid_index * max_particles_per_cell + slot_index] = i;
  }
}

__global__ void compute_density(float *x_particle, float *y_particle, float *density, int *grid_indices, int *particles_per_cell,
                                int max_particles_per_cell, int max_particles, int grid_width, int grid_height, float cell_size,
                                float kernel_radius, float kernel_vol_inv)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x; // particle id
  if (i >= max_particles)
  {
    return;
  }

  float x = x_particle[i];
  float y = y_particle[i];

  int x_cell = x / cell_size;
  int y_cell = y / cell_size;

  int x_cell_min = max(0, x_cell - 1);
  int y_cell_min = max(0, y_cell - 1);
  int x_cell_max = min(grid_width - 1, x_cell + 1);
  int y_cell_max = min(grid_height - 1, y_cell + 1);

  float density_i = 0.0f;

  // Compute density[i]
  for (int yc = y_cell_min; yc <= y_cell_max; ++yc)
  {
    for (int xc = x_cell_min; xc <= x_cell_max; ++xc)
    {
      int cell_index = yc * grid_width + xc;
      int particles_in_cell = particles_per_cell[cell_index];
      for (int slot_index = 0; slot_index < particles_in_cell; ++slot_index)
      {
        int other_i = grid_indices[slot_index + cell_index * max_particles_per_cell];
        float x_other = x_particle[other_i];
        float y_other = y_particle[other_i];

        float dx = x_other - x;
        float dy = y_other - y;
        float r = sqrtf(dx * dx + dy * dy);

        if (r >= kernel_radius)
          continue;

        float kernel_value = sharp_kernel(r, kernel_radius, kernel_vol_inv);
        density_i += PARTICLE_MASS * kernel_value;
      }
    }
  }

  // Assign computed density
  density[i] = density_i;
}

__global__ void compute_forces(float *x_particle, float *y_particle, float *x_velocity, float *y_velocity,
                               float *x_acceleration, float *y_acceleration, int *grid_indices, int *particles_per_cell, int max_particles_per_cell,
                               float *density, int max_particles, int grid_width, int grid_height, float cell_size,
                               float kernel_radius, float kernel_vol_inv)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x; // particle id
  if (i >= max_particles)
  {
    return;
  }

  float x = x_particle[i];
  float y = y_particle[i];

  int x_cell = x / cell_size;
  int y_cell = y / cell_size;

  int x_cell_min = max(0, x_cell - 1);
  int y_cell_min = max(0, y_cell - 1);
  int x_cell_max = min(grid_width - 1, x_cell + 1);
  int y_cell_max = min(grid_height - 1, y_cell + 1);

  float density_i = density[i];
  float pressure_i = (density_i - TARGET_PRESSURE) * PRESSURE_MULTIPLIER;

  // Compute pressure gradient and viscosity forces
  float pressure_grad_x = 0.0f;
  float pressure_grad_y = 0.0f;
  float viscosity_force_x = 0.0f;
  float viscosity_force_y = 0.0f;

  for (int yc = y_cell_min; yc <= y_cell_max; ++yc)
  {
    for (int xc = x_cell_min; xc <= x_cell_max; ++xc)
    {
      int cell_index = yc * grid_width + xc;
      int particles_in_cell = particles_per_cell[cell_index];
      for (int slot_index = 0; slot_index < particles_in_cell; ++slot_index)
      {
        int other_i = grid_indices[slot_index + cell_index * max_particles_per_cell];
        if (other_i == i)
          continue;

        float x_other = x_particle[other_i];
        float y_other = y_particle[other_i];

        float dx = x_other - x;
        float dy = y_other - y;
        float r = sqrtf(dx * dx + dy * dy);

        if (r >= kernel_radius || r < 1e-6f)
          continue;

        float dir_x = dx / r;
        float dir_y = dy / r;

        float density_j = density[other_i];
        float pressure_j = (density_j - TARGET_PRESSURE) * PRESSURE_MULTIPLIER;
        float shared_pressure = (pressure_i + pressure_j) * 0.5f;
        float kernel_derivative = sharp_kernel_derivative(r, kernel_radius, kernel_vol_inv);

        pressure_grad_x += PARTICLE_MASS * shared_pressure * kernel_derivative * dir_x / density_j;
        pressure_grad_y += PARTICLE_MASS * shared_pressure * kernel_derivative * dir_y / density_j;

        float influence = smoothstep_kernel(r, kernel_radius, kernel_vol_inv);
        float vx_i = x_velocity[i];
        float vy_i = y_velocity[i];
        float vx_j = x_velocity[other_i];
        float vy_j = y_velocity[other_i];

        viscosity_force_x += (vx_j - vx_i) * influence;
        viscosity_force_y += (vy_j - vy_i) * influence;
      }
    }
  }

  // Compute total acceleration
  float acc_x = -pressure_grad_x / density_i + VISCOSITY_MULTIPLIER * viscosity_force_x / density_i;
  float acc_y = -pressure_grad_y / density_i + VISCOSITY_MULTIPLIER * viscosity_force_y / density_i + GRAVITY_ACCELERATION;

  // Wall accelerations
  if (x < 0.0f)
  {
    acc_x += WALL_ACCEL_PER_DIST * -x;
  }
  else if (x > (grid_width * cell_size))
  {
    acc_x += WALL_ACCEL_PER_DIST * ((grid_width * cell_size) - x);
  }
  if (y < 0.0f)
  {
    acc_y += WALL_ACCEL_PER_DIST * -y;
  }
  else if (y > (grid_height * cell_size))
  {
    acc_y += WALL_ACCEL_PER_DIST * ((grid_height * cell_size) - y);
  }

  // Write accelerations
  x_acceleration[i] = acc_x;
  y_acceleration[i] = acc_y;
}

__global__ void update_positions_velocities(float *x_particle, float *y_particle, float *x_velocity, float *y_velocity,
                                            float *x_acceleration, float *y_acceleration, int max_particles,
                                            float dt, float bounds_x, float bounds_y)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x; // particle id
  if (i >= max_particles)
  {
    return;
  }

  x_velocity[i] += dt * x_acceleration[i];
  y_velocity[i] += dt * y_acceleration[i];

  x_particle[i] += dt * x_velocity[i];
  y_particle[i] += dt * y_velocity[i];

  // Wrap positions (assuming periodic boundary conditions)
  if (x_particle[i] < 0.0f)
  {
    x_particle[i] += bounds_x;
  }
  else if (x_particle[i] >= bounds_x)
  {
    x_particle[i] -= bounds_x;
  }

  if (y_particle[i] < 0.0f)
  {
    y_particle[i] += bounds_y;
  }
  else if (y_particle[i] >= bounds_y)
  {
    y_particle[i] -= bounds_y;
  }
}

namespace particles
{

void update() {
  
}

}