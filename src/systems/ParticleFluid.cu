#include "hip/hip_runtime.h"
#include "ParticleFluid.cuh"

#include <random>
#include <iostream>

#include "imgui.h"

#include "Kernels.cuh"

constexpr float KERNEL_RADIUS = 16.0f;
constexpr float CELL_SIZE = KERNEL_RADIUS * 1;
constexpr int PARTICLES_PER_CELL = 64;

__host__ __device__ int particle_to_cid(float x, float y, int grid_width)
{
  int grid_x = x / CELL_SIZE;
  int grid_y = y / CELL_SIZE;
  return grid_y * grid_width + grid_x;
}

__global__ void reset_particles_per_cell(int *particles_per_cell, int grid_size)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x; // grid index
  if (i < grid_size)
  {
    particles_per_cell[i] = 0;
  }
}

__global__ void populate_grid_indices(float *x_particle, float *y_particle, int *grid_indices, int max_particles,
                                      int *particles_per_cell, int max_particles_per_cell,
                                      int grid_width, int grid_height)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x; // particle id
  if (i >= max_particles)
  {
    return;
  }

  float x = x_particle[i];
  float y = y_particle[i];

  int grid_index = particle_to_cid(x, y, grid_width);

  int slot_index = atomicAdd(&particles_per_cell[grid_index], 1);
  if (slot_index < max_particles_per_cell)
  {
    grid_indices[grid_index * max_particles_per_cell + slot_index] = i;
  }
  else
  {
    // atomicSub(&particles_per_cell[grid_index], 1);
    // TODO: mark this particle as not in a cell and move it somewhere else
    // TODO: implement near field repulsion to see if it fixes this issue
  }
}

__global__ void compute_density(float *x_particle, float *y_particle, float *density, int *grid_indices, int *particles_per_cell,
                                int max_particles_per_cell, int max_particles, int grid_width, int grid_height, float kernel_vol_inv,
                                particles::TunableParams params)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x; // particle id
  if (i >= max_particles)
  {
    return;
  }

  float x = x_particle[i];
  float y = y_particle[i];

  int x_cell = x / CELL_SIZE;
  int y_cell = y / CELL_SIZE;

  int x_cell_min = max(0, x_cell - 1);
  int y_cell_min = max(0, y_cell - 1);
  int x_cell_max = min(grid_width - 1, x_cell + 1);
  int y_cell_max = min(grid_height - 1, y_cell + 1);

  float density_i = 0.0f;

  // Compute density[i]
  for (int yc = y_cell_min; yc <= y_cell_max; ++yc)
  {
    for (int xc = x_cell_min; xc <= x_cell_max; ++xc)
    {
      int cell_index = yc * grid_width + xc;
      int particles_in_cell = particles_per_cell[cell_index];
      for (int slot_index = 0; slot_index < particles_in_cell; ++slot_index)
      {
        int other_i = grid_indices[slot_index + cell_index * max_particles_per_cell];
        float x_other = x_particle[other_i];
        float y_other = y_particle[other_i];

        float dx = x_other - x;
        float dy = y_other - y;
        // TODO: wrap around for periodic boundary conditions (horizontal only)
        float r = sqrtf(dx * dx + dy * dy);

        if (r >= KERNEL_RADIUS)
          continue;

        float kernel_value = sharp_kernel(r, KERNEL_RADIUS, kernel_vol_inv);
        density_i += params.particle_mass * kernel_value;
      }
    }
  }

  // Assign computed density
  density[i] = density_i;
}

__global__ void compute_forces(float *x_particle, float *y_particle, float *x_velocity, float *y_velocity,
                               float *x_acceleration, float *y_acceleration, int *grid_indices, int *particles_per_cell, int max_particles_per_cell,
                               float *density, int max_particles, int grid_width, int grid_height,
                               float smoothkernel_vol_inv, float sharpkernel_vol_inv, particles::TunableParams params)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x; // particle id
  if (i >= max_particles)
  {
    return;
  }

  float x = x_particle[i];
  float y = y_particle[i];

  int x_cell = x / CELL_SIZE;
  int y_cell = y / CELL_SIZE;

  int x_cell_min = max(0, x_cell - 1);
  int y_cell_min = max(0, y_cell - 1);
  int x_cell_max = min(grid_width - 1, x_cell + 1);
  int y_cell_max = min(grid_height - 1, y_cell + 1);

  float density_i = density[i];
  float pressure_i = (density_i - params.target_pressure) * params.pressure_multiplier;

  // Compute pressure gradient and viscosity forces
  float pressure_grad_x = 0.0f;
  float pressure_grad_y = 0.0f;
  float viscosity_force_x = 0.0f;
  float viscosity_force_y = 0.0f;

  for (int yc = y_cell_min; yc <= y_cell_max; ++yc)
  {
    for (int xc = x_cell_min; xc <= x_cell_max; ++xc)
    {
      int cell_index = yc * grid_width + xc;
      int particles_in_cell = particles_per_cell[cell_index];
      for (int slot_index = 0; slot_index < particles_in_cell; ++slot_index)
      {
        int other_i = grid_indices[slot_index + cell_index * max_particles_per_cell];
        if (other_i == i)
          continue;

        float x_other = x_particle[other_i];
        float y_other = y_particle[other_i];

        float dx = x_other - x;
        float dy = y_other - y;
        float r = sqrtf(dx * dx + dy * dy);

        if (r >= KERNEL_RADIUS || r < 1e-6f)
          continue;

        float dir_x = dx / r;
        float dir_y = dy / r;

        float density_j = density[other_i];
        float pressure_j = (density_j - params.target_pressure) * params.pressure_multiplier;
        float shared_pressure = (pressure_i + pressure_j) * 0.5f;
        float kernel_derivative = sharp_kernel_derivative(r, KERNEL_RADIUS, sharpkernel_vol_inv);

        pressure_grad_x += params.particle_mass * shared_pressure * kernel_derivative * dir_x / density_j;
        pressure_grad_y += params.particle_mass * shared_pressure * kernel_derivative * dir_y / density_j;

        float influence = smoothstep_kernel(r, KERNEL_RADIUS, smoothkernel_vol_inv);
        float vx_i = x_velocity[i];
        float vy_i = y_velocity[i];
        float vx_j = x_velocity[other_i];
        float vy_j = y_velocity[other_i];

        viscosity_force_x += (vx_j - vx_i) * influence;
        viscosity_force_y += (vy_j - vy_i) * influence;
      }
    }
  }

  // Compute total acceleration
  float acc_x = -pressure_grad_x / density_i + params.viscosity_multiplier * viscosity_force_x / density_i;
  float acc_y = -pressure_grad_y / density_i + params.viscosity_multiplier * viscosity_force_y / density_i + params.gravity_acceleration;

  // Write accelerations
  x_acceleration[i] = acc_x;
  y_acceleration[i] = acc_y;
}

__global__ void update_positions_velocities(float *x_particle, float *y_particle, float *x_velocity, float *y_velocity,
                                            float *x_acceleration, float *y_acceleration, int max_particles,
                                            float dt, float bounds_x, float bounds_y, particles::TunableParams params)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x; // particle id
  if (i >= max_particles)
  {
    return;
  }

  // calculate drag
  float x_drag = -x_velocity[i] * abs(x_velocity[i]) * params.drag;
  float y_drag = -y_velocity[i] * abs(y_velocity[i]) * params.drag;

  x_velocity[i] += dt * (x_acceleration[i] + x_drag);
  y_velocity[i] += dt * (y_acceleration[i] + y_drag);

  x_particle[i] += dt * x_velocity[i];
  y_particle[i] += dt * y_velocity[i];

  // Bounce off walls
  if (x_particle[i] < 0.0f)
  {
    x_velocity[i] = abs(x_velocity[i]);
    x_particle[i] = 0.0f;
  }
  else if (x_particle[i] >= bounds_x)
  {
    x_velocity[i] = -abs(x_velocity[i]);
    x_particle[i] = bounds_x - 1e-3f; // TODO: i think there is a way to get the exact epsilon
  }

  if (y_particle[i] < 0.0f)
  {
    y_velocity[i] = abs(y_velocity[i]);
    y_particle[i] = 0.0f;
  }
  else if (y_particle[i] >= bounds_y)
  {
    y_velocity[i] = -abs(y_velocity[i]);
    y_particle[i] = bounds_y - 1e-4f;
  }
}

__global__ void render_kernel(unsigned int *circle_vbo, float *x, float *y, float radius, size_t count)
{
  // circle_vbo format is (float x, float y, float radius, unsigned int color)
  // for now, hard code color to white
  unsigned int color = 0xFFFFFFFF;
  size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < count)
  {
    circle_vbo[i * 4 + 0] = reinterpret_cast<unsigned int &>(x[i]);
    circle_vbo[i * 4 + 1] = reinterpret_cast<unsigned int &>(y[i]);
    circle_vbo[i * 4 + 2] = reinterpret_cast<unsigned int &>(radius);
    circle_vbo[i * 4 + 3] = color;
  }
}

namespace particles
{
  ParticleFluid::ParticleFluid(int width, int height, bool use_graphics)
  {
    if (use_graphics)
    {
      circle_renderer = std::make_unique<CircleRenderer>();
    }
    // configure grid
    grid.reconfigure(width / 2, height / 2, PARTICLES_PER_CELL);
    // temp: init some particles in the bottom half
    std::default_random_engine rand;
    std::uniform_real_distribution<float> dist_x(0.0f, width * 8 * 0.75f); // TODO: 8 is the soil cell size
    std::uniform_real_distribution<float> dist_y(0.0f, height * 8 * 0.75f);

    // velocity init with gaussian
    std::normal_distribution<float> dist_vel(0.0f, 1.0f);

    // temp: 1000 particles
    constexpr int NUM_PARTICLES = 40000;
    particles.resize_all(NUM_PARTICLES);
    for (int i = 0; i < NUM_PARTICLES; ++i)
    {
      particles.x_particle[i] = dist_x(rand);
    }
    for (int i = 0; i < NUM_PARTICLES; ++i)
    {
      particles.y_particle[i] = dist_y(rand);
    }
    for (int i = 0; i < NUM_PARTICLES; ++i)
    {
      particles.x_velocity[i] = dist_vel(rand);
    }
    for (int i = 0; i < NUM_PARTICLES; ++i)
    {
      particles.y_velocity[i] = dist_vel(rand);
    }
    // accel is defaulted to 0
    // density is defaulted to 0
    // there are some vectors that are present in the C++ host version, but missing in the CUDA version.
    // TODO: verify correctness of algo

    // send to device
    particles_device.copy_from_host(particles);
    grid_device.copy_from_host(grid);
  }

  void ParticleFluid::update(float dt)
  {
    // reset particles per cell
    const int grid_size = grid.width * grid.height;
    dim3 block(256);
    dim3 grid_dim((grid_size + block.x - 1) / block.x);
    reset_particles_per_cell<<<grid_dim, block>>>(grid_device.particles_per_cell.data().get(), grid_size);

    // populate grid indices
    const int particle_count = particles_device.x_particle.size();
    block = dim3(256);
    grid_dim = dim3((particle_count + block.x - 1) / block.x);
    populate_grid_indices<<<grid_dim, block>>>(particles_device.x_particle.data().get(), particles_device.y_particle.data().get(),
                                               grid_device.grid_indices.data().get(), particle_count, grid_device.particles_per_cell.data().get(),
                                               grid.max_particles_per_cell, grid.width, grid.height);

    // compute density
    // density is computed using sharp kernel
    const float sharp_kernel_vol_inv = 1.0f / sharp_kernel_volume(KERNEL_RADIUS);

    block = dim3(256);
    grid_dim = dim3((particle_count + block.x - 1) / block.x);
    compute_density<<<grid_dim, block>>>(particles_device.x_particle.data().get(), particles_device.y_particle.data().get(),
                                         particles_device.density.data().get(), grid_device.grid_indices.data().get(),
                                         grid_device.particles_per_cell.data().get(), grid.max_particles_per_cell, particle_count,
                                         grid.width, grid.height, sharp_kernel_vol_inv, params);

    // compute forces
    // compute_forces uses sharp kernel for pressure and smooth kernel for viscosity
    const float smooth_kernel_vol_inv = 1.0f / smoothstep_kernel_volume(KERNEL_RADIUS);
    block = dim3(256);
    grid_dim = dim3((particle_count + block.x - 1) / block.x);
    compute_forces<<<grid_dim, block>>>(particles_device.x_particle.data().get(), particles_device.y_particle.data().get(),
                                        particles_device.x_velocity.data().get(), particles_device.y_velocity.data().get(),
                                        particles_device.x_accel.data().get(), particles_device.y_accel.data().get(),
                                        grid_device.grid_indices.data().get(), grid_device.particles_per_cell.data().get(),
                                        grid.max_particles_per_cell, particles_device.density.data().get(), particle_count,
                                        grid.width, grid.height, smooth_kernel_vol_inv, sharp_kernel_vol_inv, params);

    // update positions and velocities
    const float bounds_x = grid.width * CELL_SIZE;
    const float bounds_y = grid.height * CELL_SIZE;
    block = dim3(256);
    grid_dim = dim3((particle_count + block.x - 1) / block.x);
    update_positions_velocities<<<grid_dim, block>>>(particles_device.x_particle.data().get(), particles_device.y_particle.data().get(),
                                                     particles_device.x_velocity.data().get(), particles_device.y_velocity.data().get(),
                                                     particles_device.x_accel.data().get(), particles_device.y_accel.data().get(),
                                                     particle_count, dt, bounds_x, bounds_y, params);
  }

  void ParticleFluid::render(const glm::mat4 &transform)
  {
    // early return if we don't have a renderer
    if (!circle_renderer)
      return;

    // float pressure_multiplier = 4000.0f;
    // float viscosity_multiplier = 2.0f;
    // float target_pressure = 0.6f;
    // float particle_mass = 1.0f;
    // float gravity_acceleration = -30.0f;
    // float drag = 0.000f;

    // connect tunable params to imgui
    ImGui::Begin("Particle Fluid");
    ImGui::SliderFloat("Pressure Multiplier", &params.pressure_multiplier, 0.0f, 10000.0f);
    ImGui::SliderFloat("Viscosity Multiplier", &params.viscosity_multiplier, 0.0f, 32.0f);
    ImGui::SliderFloat("Target Pressure", &params.target_pressure, 0.0f, 4.0f);
    ImGui::SliderFloat("Particle Mass", &params.particle_mass, 0.0f, 4.0f);
    ImGui::SliderFloat("Gravity Acceleration", &params.gravity_acceleration, -200.0f, 0.0f);
    ImGui::SliderFloat("Drag", &params.drag, 0.0f, 0.01f);
    ImGui::End();

    circle_renderer->set_transform(transform);

    const auto circle_count = particles_device.x_particle.size();
    circle_renderer->ensure_vbo_capacity(circle_count);
    // get a cuda compatible pointer to the vbo
    circle_renderer->cuda_register_buffer();
    auto vbo_ptr = circle_renderer->cuda_map_buffer();
    // render the particles
    dim3 block(256);
    dim3 grid_dim((circle_count + block.x - 1) / block.x);

    render_kernel<<<grid_dim, block>>>(static_cast<unsigned int *>(vbo_ptr), particles_device.x_particle.data().get(),
                                       particles_device.y_particle.data().get(), KERNEL_RADIUS, circle_count);
    // unmap the buffer
    circle_renderer->cuda_unmap_buffer();
    // render the circles
    circle_renderer->render(circle_count);
    circle_renderer->cuda_unregister_buffer(); // TODO: is this necessary?
  }

}